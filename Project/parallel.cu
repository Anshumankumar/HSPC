#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include "graph.h"

__global__ void travese(int *array, int *xadj, int *adj,
        int *current, int *Fs, int *entry)
{
    array[*current] = array[*current] +1;
}

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        std::cout << "Invalid no of argument\n";
        std::cout << "Usage: graphTraversal NoOfVertex density\n";
        exit(-2);
    }
    std::vector<int>array,xadj,adj,color;
    int n = atoi(argv[1]);
    int density = atoi(argv[2]);
    std::cout << "No of Vertices: " << n << "\ndensity No of Connection: "
        << density << "\n";

    graphGenerator(n,density, array,xadj,adj);
    int * dev_array;
    int *dev_xadj;
    int *dev_adj;
    hipMalloc((void**)&dev_array, sizeof(int)*array.size());
    hipMalloc((void**)&dev_xadj, sizeof(int)*xadj.size());
    hipMalloc((void**)&dev_adj, sizeof(int)*adj.size());

    hipMemcpy( dev_array, &array[0],
            sizeof(int)*array.size(), hipMemcpyHostToDevice );

    hipMemcpy( dev_xadj, &xadj[0],
            sizeof(int)*xadj.size(), hipMemcpyHostToDevice );

    hipMemcpy( dev_adj, &adj[0],
            sizeof(int)*adj.size(), hipMemcpyHostToDevice );


    hipMemcpy(&array[0], dev_array,
            sizeof(int)*array.size(), hipMemcpyHostToDevice );
    int stop =1;
    int start = 0;
    matrixMultiplication<<<stop-start,1>>>
    std::cout << dev_array[n-1] <<"\n";
#ifdef DEBUG
    print(xadj);
    print(adj);
#endif
    return 0;
}
