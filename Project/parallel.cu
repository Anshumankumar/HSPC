#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include "graph.h"

__global__ void traverse(int *array, int *xadj, int *adj,
        int *current, int *fs, int *entry)
{
    int index = fs[(*current)+blockIdx.x];
    if (index != 0)
    {
        if (array[index] ==  -1)
        {
            for (int i=xadj[index-1]; i < xadj[index]; i++)
                fs[atomicAdd(entry,1)] = adj[i];
            array[index] = 0;
        }
    }
}

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        std::cout << "Invalid no of argument\n";
        std::cout << "Usage: graphTraversal NoOfVertex density\n";
        exit(-2);
    }
    std::vector<int>array,xadj,adj,fs;
    int n = atoi(argv[1]);
    int density = atoi(argv[2]);
    std::cout << "No of Vertices: " << n << "\ndensity No of Connection: "
        << density << "\n";

    graphGenerator(n,density, array,xadj,adj);

    print(array);
    int * dev_array;
    int *dev_xadj;
    int *dev_adj;
    int *dev_entry;
    int *dev_fs;
    int *dev_current;
    hipMalloc((void**)&dev_array, sizeof(int)*array.size());
    hipMalloc((void**)&dev_xadj, sizeof(int)*xadj.size());
    hipMalloc((void**)&dev_adj, sizeof(int)*adj.size());
    hipMalloc((void**)&dev_fs, sizeof(int)*adj.size()*4);
    hipMalloc((void**)&dev_entry, sizeof(int));
    hipMalloc((void**)&dev_current, sizeof(int));
    hipMemcpy( dev_array, &array[0],
            sizeof(int)*array.size(), hipMemcpyHostToDevice );

    hipMemcpy( dev_xadj, &xadj[0],
            sizeof(int)*xadj.size(), hipMemcpyHostToDevice );

    hipMemcpy( dev_adj, &adj[0],
            sizeof(int)*adj.size(), hipMemcpyHostToDevice );


    fs.resize(4*adj.size());
    fs[0] = n-1;
    hipMemcpy( dev_fs, &fs[0],
            sizeof(int)*fs.size(), hipMemcpyHostToDevice );

    int stop =1;
    int start = 0;

    while(start != stop)
    {
        hipMemcpy( dev_current, &start,
                sizeof(int), hipMemcpyHostToDevice );
        hipMemcpy( dev_entry, &stop,
                sizeof(int), hipMemcpyHostToDevice );
        traverse<<<stop-start,1>>>(dev_array, dev_xadj, dev_adj,
                dev_current, dev_fs, dev_entry);
        start = stop;
        hipMemcpy( &stop, dev_entry,
                sizeof(int), hipMemcpyDeviceToHost );

    }
    std::cout << "SUCCESSFUL TILL HERE \n";
    hipMemcpy(&array[0], dev_array,
            sizeof(int)*array.size(), hipMemcpyDeviceToHost );
    hipMemcpy(&fs[0], dev_fs,
            sizeof(int)*fs.size(), hipMemcpyDeviceToHost );




#ifdef DEBUG
    print(fs);
    print(array);
    print(xadj);
    print(adj);
#endif
    return 0;
}
