#include "hip/hip_runtime.h"
#include "graph.h"


void print(std::vector<int> a)
{
    for(int i=0; i<a.size(); i++)
    {
        std::cout <<a[i] << " ";
    }
    std::cout << "\n";
}

void graphGenerator(int n, int maxCon, std::vector<int> &a,
        std::vector<int> &xadj, std::vector<int> &adj)
{
    srand(time(NULL));
    int noOfEdges = 0;
    for (int i=0;i<n;i++)
    {
        if((i%100)==0) std::cout << i <<"\n";
        a.push_back(rand()%1000);
        int cEdge = std::min(i+1, maxCon);
        cEdge = 1 + rand()%cEdge;
        if (cEdge>=i) cEdge = i;

        std::vector<int> currentList;
        for (int j=0; j< cEdge;)
        {
            int num = rand()%(i+1);
            if (num >=i) continue;
            std::vector<int>::iterator it;
            it = find(currentList.begin(),currentList.end(),num);
            if (it == currentList.end())
            {
                currentList.push_back(num);
                j++;
            }
        }

        noOfEdges += cEdge;
        cEdge = rand()%maxCon;
        for (int j=0; j< cEdge;)
        {
            int num = rand()%n;
            std::vector<int>::iterator it;
            it = find(currentList.begin(),currentList.end(),num);
            if (it == currentList.end())
            {
                currentList.push_back(num);
                j++;
            }
        }

        for(int i=0;i<currentList.size();i++)
        {
            adj.push_back(currentList[i]);
        }

        noOfEdges += cEdge;
        xadj.push_back(noOfEdges);
    }
}

